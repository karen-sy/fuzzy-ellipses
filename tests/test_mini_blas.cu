#include "../fmb/blas_primitives.h"

// Helper function to check CUDA errors
void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void matmul_cpu_naive(
    int32_t size_i,
    int32_t size_j,
    int32_t size_k,
    float const *a,
    float const *b,
    float *c) {
    for (int32_t i = 0; i < size_i; ++i) {
        for (int32_t j = 0; j < size_j; ++j) {
            float sum = 0.0;
            for (int32_t k = 0; k < size_k; ++k) {
                sum += a[i * size_k + k] * b[k * size_j + j];
            }
            c[i * size_j + j] = sum;
        }
    }
}

int main() {
    // Dimensions of the matrices
    int M = 3; // Rows of A and C
    int N = 4; // Columns of B and C
    int K = 3; // Columns of A, Rows of B

    // Allocate and initialize host matrices in row-major order
    float h_A[M * K] = { 1, 2, 3, 4, 5, 6, 7, 8 };
    float h_B[K * N] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12 };
    float h_C[N * M] = { 0 }; // Result matrix
    float h_C_host[M * N] = { 0 }; // Result matrix

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc((void**)&d_A, M * K * sizeof(float)), "Failed to allocate device memory for A");
    checkCudaError(hipMalloc((void**)&d_B, K * N * sizeof(float)), "Failed to allocate device memory for B");
    checkCudaError(hipMalloc((void**)&d_C, M * N * sizeof(float)), "Failed to allocate device memory for C");

    // Copy host matrices to device
    checkCudaError(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice), "Failed to copy A to device");
    checkCudaError(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice), "Failed to copy B to device");

    // run cuBLAS matmul
    blas::matmul(M, N, K, d_A, d_B, d_C);
    blas::matmul(M, N, K, d_A, d_B, d_C); // run twice to make sure no cublas session issues

    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost), "Failed to copy C to host");

    std::cout << "Result matrix C:" << std::endl;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Compare with CPU result
    matmul_cpu_naive(M, N, K, h_A, h_B, h_C_host);
    std::cout << "Reference matrix C:" << std::endl;
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_C_host[i * N + j] << " ";
        }
        std::cout << std::endl;
    }


    // Cleanup
    // hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
